#include "hip/hip_runtime.h"
#include <chrono>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <>

#include <stdio.h>

#include "boruvkas.h"

// Define constants for CUDA threadblocks
#define THREADBLOCK_WIDTH (8)
#define THREADBLOCK_HEIGHT (8)
#define BLOCKSIZE (THREADBLOCK_WIDTH*THREADBLOCK_HEIGHT)

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {
    // TODO
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;


__global__ void test(int *a) {
    int threadID = threadIdx.y * blockDim.x + threadIdx.x;
    a[threadID] = threadID;
}

inline int get_component(std::vector<Vertex>& componentlist, const int i) {
    int curr = componentlist[i].component;

    while (componentlist[curr].component != curr) {
        curr = componentlist[curr].component;
    }

    componentlist[i].component = curr;
    return curr;
}

inline void merge_components(std::vector<Vertex>& componentlist, const int i, const int j) {
    componentlist[get_component(componentlist, i)].component = get_component(componentlist, j);
}

std::vector<Edge>* boruvka_mst_helper(int n_vertices, const std::vector<Edge>& edgelist) {
    std::vector<Edge>* mst = new std::vector<Edge>();
    std::vector<Vertex> vertices(n_vertices);

    // initialize components
    for (int i = 0; i < n_vertices; i++) {
        vertices[i] = Vertex{i, nullptr}; //Vertex{i, i, nullptr};
    }

    int n_components = n_vertices;
    bool keep_going;

    do {
        keep_going = false;
        for (const Edge& e : edgelist) {
            int c1 = get_component(vertices, e.u);
            int c2 = get_component(vertices, e.v);

            // Skip edges that connect a component to itself
            if (c1 == c2) {
                continue;
            }

            // Check if this edge is the cheapest (so far) for its connected components
            if (vertices[c1].cheapest_edge == nullptr || e < *vertices[c1].cheapest_edge) {
                vertices[c1].cheapest_edge = &e;
            }
            if (vertices[c2].cheapest_edge == nullptr || e < *vertices[c2].cheapest_edge) {
                vertices[c2].cheapest_edge = &e;
            }
        }

        keep_going = false;
        // Connect newest edges to MST
        for (int i = 0; i < n_vertices; i++) {
            const Edge* edge_ptr = vertices[i].cheapest_edge;
            if (edge_ptr == nullptr) {
                continue;
            }

            // if (get_component(vertices, edge_ptr->u) == get_component(vertices, edge_ptr->v)) {
            //     continue;
            // }

            mst->push_back(*edge_ptr);
            vertices[get_component(vertices, edge_ptr->u)].cheapest_edge = nullptr;
            vertices[get_component(vertices, edge_ptr->v)].cheapest_edge = nullptr;
            merge_components(vertices, edge_ptr->u, edge_ptr->v);
            n_components--;
            keep_going = true;
        }

    } while (keep_going && n_components > 1);

    return mst;
}

MST boruvka_mst(int n_vertices, const std::vector<Edge>& edgelist) {
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for Parallel Boruvka's\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("NVIDIA GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }

    // TODO: all this code is silly, just tests that we have CUDA set up correctly
    {
        int a[BLOCKSIZE] = {0};
        int *d_a;

        // Allocate device memory for a
        hipMalloc((void**)&d_a, sizeof(int) * BLOCKSIZE);


        for (int i = 0; i < BLOCKSIZE; i++) {
            a[i] = 0;
        }
        for (int i = 0; i < BLOCKSIZE; i++) {
            printf("%d ", a[i]);
        }
        printf("\n");


        // Transfer data from host to device memory
        hipMemcpy(d_a, a, sizeof(int) * BLOCKSIZE, hipMemcpyHostToDevice);

        test<<<1, BLOCKSIZE>>>(d_a);

        // Transfer data from device to host memory
        hipMemcpy(a, d_a, sizeof(int) * BLOCKSIZE, hipMemcpyDeviceToHost);

        for (int i = 0; i < BLOCKSIZE; i++) {
            printf("%d ", a[i]);
        }
        printf("\n");

        hipFree(d_a);
    }

    MST result;
    result.mst = boruvka_mst_helper(n_vertices, edgelist);

    result.weight = 0;
    for (const Edge& e : *result.mst) {
        result.weight += e.weight;
    }

    return result;
}
