#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <driver_functions.h>
#include <limits.h>
#include <stdio.h>

#include <chrono>
#include <string>

#include "boruvkas.h"

// Define constants for CUDA threadblocks
#define BLOCKSIZE (1024)

#define NBLOCKS_ASSIGN_CHEAPEST (128)
#define NBLOCKS_OTHER (128)

#define NTHREADS_ASSIGN_CHEAPEST (NBLOCKS_ASSIGN_CHEAPEST * BLOCKSIZE)
#define NTHREADS_OTHER (NBLOCKS_OTHER * BLOCKSIZE)

#define NO_EDGE (ULONG_MAX)

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {
    Vertex* vertices;
    Edge* edges;
    char* mst_tree;
    ullong n_vertices;
    ullong n_edges;
};

// Another global value
__device__ ullong n_unions_total;

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstGraphParams;

__device__ inline int edge_cmp(const Edge* edges, const ullong i, const ullong j)
{
    if (i == j) return 0;

    const Edge& lhs = edges[i];
    const Edge& rhs = edges[j];

    if (lhs.weight < rhs.weight) {
        return -1;
    }
    if (lhs.weight > rhs.weight) {
        return 1;
    }

    if (i < j) {
        return -1;
    }
    return 1;
}

__device__ inline ullong get_component(Vertex* componentlist, const ullong i) {
    ullong curr = componentlist[i].component;

    // while (componentlist[curr].component != curr) {
    //     curr = componentlist[curr].component;
    // }

    return curr;
}

__device__ inline void flatten_component(Vertex* componentlist, const ullong i) {
    ullong curr = componentlist[i].component;

    while (componentlist[curr].component != curr) {
        curr = componentlist[curr].component;
    }

    // Flatten component trees
    componentlist[i].component = curr;
    // if (componentlist[i].component != curr) {
    //     atomicExch(&componentlist[i].component, curr);
    // }
}


__device__ inline void merge_components(Vertex* componentlist, const ullong i,
                                        const ullong j) {
    // ullong u = i;
    // ullong v = j;
    componentlist[i].component = j;
    // const ullong v = get_component(componentlist, j);
    // ullong old;
    // do {
    //     u = get_component(componentlist, u);
    //     old = atomicCAS(&(componentlist[u].component), u, v);
    // } while (old != u);
}

__global__ void init_arrs() {
    const int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    const ullong n_vertices = cuConstGraphParams.n_vertices;
    Vertex* const vertices = cuConstGraphParams.vertices;

    const ullong start = (threadID * n_vertices / NTHREADS_OTHER);
    const ullong end = ((threadID + 1) * n_vertices / NTHREADS_OTHER);

    // initialize components
    for (ullong i = start; i < end; i++) {
        vertices[i] = Vertex{i, NO_EDGE};
    }
}

__global__ void reset_arrs() {
    const int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    const ullong n_vertices = cuConstGraphParams.n_vertices;
    Vertex* const vertices = cuConstGraphParams.vertices;

    const ullong start = (threadID * n_vertices / NTHREADS_OTHER);
    const ullong end = ((threadID + 1) * n_vertices / NTHREADS_OTHER);

    // initialize components
    for (ullong i = start; i < end; i++) {
        vertices[i].cheapest_edge = NO_EDGE;
        flatten_component(vertices, i);
    }
}

__global__ void assign_cheapest() {
    const int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    // Renaming to make life easier, this gets compiled away
    const ullong n_edges = cuConstGraphParams.n_edges;
    Vertex* const vertices = cuConstGraphParams.vertices;
    Edge* const edges = cuConstGraphParams.edges;

    const ullong start = (threadID * n_edges) / NTHREADS_ASSIGN_CHEAPEST;
    const ullong end = ((threadID + 1) * n_edges) / NTHREADS_ASSIGN_CHEAPEST;

    for (ullong i = start; i < end; i++) {
        Edge& e = edges[i];
        e.u = get_component(vertices, e.u);
        e.v = get_component(vertices, e.v);

        // Skip edges that connect a component to itself
        if (e.u == e.v) {
            continue;
        }

        // Atomic update cheapest_edge[u]
        ullong expected = vertices[e.u].cheapest_edge;
        ullong old;
        while (expected == NO_EDGE || edge_cmp(edges, i, expected) < 0) {
            old = atomicCAS(&vertices[e.u].cheapest_edge, expected, i);
            if (expected == old) {
                break;
            }
            expected = old;
        }

        // Atomic update cheapest_edge[v]
        expected = vertices[e.v].cheapest_edge;
        while (expected == NO_EDGE || edge_cmp(edges, i, expected) < 0) {
            old = atomicCAS(&vertices[e.v].cheapest_edge, expected, i);
            if (expected == old) {
                break;
            }
            expected = old;
        }
    }
}

__global__ void update_mst() {
    const int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    // Renaming to make life easier, this gets compiled away
    const ullong n_vertices = cuConstGraphParams.n_vertices;
    Vertex* const vertices = cuConstGraphParams.vertices;
    Edge* const edges = cuConstGraphParams.edges;

    const ullong start = (threadID * n_vertices) / NTHREADS_OTHER;
    const ullong end = ((threadID + 1) * n_vertices) / NTHREADS_OTHER;

    ullong n_unions_made = 0;
    // Connect newest edges to MST
    for (ullong i = start; i < end; i++) {
        const ullong edge_ind = vertices[i].cheapest_edge;

        if (edge_ind == NO_EDGE) {
            continue;
        }

        const Edge& edge_ptr = edges[edge_ind];

        // If this edge is covered twice, only union when i == u (u < v is
        // assumed)
        if (edge_ptr.v == i &&
            edge_ind == vertices[edge_ptr.u].cheapest_edge) {
            continue;
        }

        const ullong j = (i == edge_ptr.u? edge_ptr.v : edge_ptr.u); // this is the other index

        cuConstGraphParams.mst_tree[edge_ind] = 1;
        merge_components(vertices, i, j);
        n_unions_made++;
    }

    atomicAdd(&n_unions_total, n_unions_made);
}

void initGPUs() {
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for Parallel Boruvka's\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("NVIDIA GeForce RTX 2080") == 0) {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

    printf("---------------------------------------------------------\n");
    if (!isFastGPU) {
        printf(
            "WARNING: "
            "You're not running on a fast GPU, please consider using "
            "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }
}

MST boruvka_mst(const ullong n_vertices, const ullong n_edges, const Edge* edgelist) {
    MST mst;
    mst.weight = 0;
    char* mst_tree = (char*) malloc(sizeof(char) * n_edges);

    char* device_mst_tree;
    Vertex* device_vertices;
    Edge* device_edgelist;

    hipMalloc(&device_mst_tree, sizeof(char) * n_edges);
    hipMemset(device_mst_tree, 0, sizeof(char) * n_edges);

    hipMalloc(&device_vertices, sizeof(Vertex) * n_vertices);

    hipMalloc(&device_edgelist, sizeof(Edge) * n_edges);
    hipMemcpy(device_edgelist, edgelist, sizeof(Edge) * n_edges,
               hipMemcpyHostToDevice);

    GlobalConstants params;
    params.vertices = device_vertices;
    params.edges = device_edgelist;
    params.mst_tree = device_mst_tree;
    params.n_edges = n_edges;
    params.n_vertices = n_vertices;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstGraphParams), &params, sizeof(GlobalConstants));

    // Run Boruvka's in parallel
    ullong n_unions = 0;
    ullong n_unions_old;

    // Initialise global
    hipMemcpyToSymbol(HIP_SYMBOL(n_unions_total), &n_unions, sizeof(ullong));

    init_arrs<<<NBLOCKS_OTHER, BLOCKSIZE>>>();

    do {
        n_unions_old = n_unions;

        reset_arrs<<<NBLOCKS_OTHER, BLOCKSIZE>>>();
        assign_cheapest<<<NBLOCKS_ASSIGN_CHEAPEST, BLOCKSIZE>>>();
        update_mst<<<NBLOCKS_OTHER, BLOCKSIZE>>>();
        hipMemcpyFromSymbol(&n_unions, HIP_SYMBOL(n_unions_total), sizeof(ullong));

        // debug
        // Vertex * ts = (Vertex *) malloc(sizeof(Vertex) * n_vertices);
        // hipMemcpy(ts, device_vertices, sizeof(Vertex) * n_vertices,
        // hipMemcpyDeviceToHost); for (int i = 0; i < n_vertices; i++) {
        //     printf("%d ", ts[i].cheapest_edge);
        // }
        // printf("\n");
        // Edge * ed = (Edge *) malloc(sizeof(Edge) * n_vertices);
        // hipMemcpy(ed, device_edgelist, sizeof(Edge) * n_edges,
        // hipMemcpyDeviceToHost); for (int i = 0; i < n_edges; i++) {
        //     printf("%d-%d-%d ", ed[i].u, ed[i].v, ed[i].weight);
        // }
        // printf("nc %d\n", n_comp);
    } while (n_unions != n_unions_old && n_unions < n_vertices - 1);

    // Copy run results off of device
    hipMemcpy(mst_tree, device_mst_tree, sizeof(char) * n_edges,
               hipMemcpyDeviceToHost);
    mst.mst = mst_tree;

    // Clean up device memory
    hipFree(device_mst_tree);
    hipFree(device_vertices);
    hipFree(device_edgelist);

    // TODO: Move this into the kernel (filtering vertices to get a short list)
    // Compute final weight
    for (ullong i = 0; i < n_edges; i++) {
        if (mst.mst[i]) {
            const Edge& e = edgelist[i];
            mst.weight += e.weight;
        }
    }

    return mst;
}
